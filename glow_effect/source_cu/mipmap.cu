#include "hip/hip_runtime.h"
﻿/*******************************************************************************************************************
 * FILE NAME   :    mipmap.cu
 *
 * PROJECT NAME:    Cuda Learning
 *
 * DESCRIPTION :    mipmap genernator and access
 *
 * VERSION HISTORY
 * YYYY/MMM/DD      Author          Comments
 * 2022 OCT 10      Yu Liu          Creation
 * 2022 OCT 26      Yu Liu          Moved V-shaped curve into cuda
 * 2022 OCT 27      Yu Liu          Proved texReadMode = hipReadModeNormalizedFloat to be a must for linear filter
 *                                  also corrected phase shift by using x+1.f/y+1.f rather than x+0.5/y+0.5
 *
 ********************************************************************************************************************/
#include "old_movies.cuh"
#include "./mipmap.cuh"
extern bool button_State[5];

/**
 * CUDA kernel to generate a mipmap level by downscaling an input texture.
 *
 * This kernel performs 2x2 averaging to create a lower resolution mipmap
 * from the provided input texture and writes the results to an output surface.
 *
 * @param mipOutput  CUDA surface object for the output mipmap.
 * @param mipInput   CUDA texture object for the input image.
 * @param imageW     Width of the output mipmap level.
 * @param imageH     Height of the output mipmap level.
 */
__global__ void d_gen_mipmap(
	hipSurfaceObject_t mipOutput,
	hipTextureObject_t mipInput,
	uint imageW,
	uint imageH
) {
	// Compute the thread's x and y coordinates within the grid.
	uint x = blockIdx.x * blockDim.x + threadIdx.x;
	uint y = blockIdx.y * blockDim.y + threadIdx.y;

	// Compute the normalized pixel width and height.
	float px = 1.0f / static_cast<float>(imageW);
	float py = 1.0f / static_cast<float>(imageH);

	// Ensure the thread operates only within valid image dimensions.
	if ((x < imageW) && (y < imageH)) {
		// Fetch and average the colors of 4 neighboring texels.
		// tex2D samples texture at normalized coordinates [0,1] range.
		float4 color =
			tex2D<float4>(mipInput, (x + 0.0f) * px, (y + 0.0f) * py) +
			tex2D<float4>(mipInput, (x + 1.0f) * px, (y + 0.0f) * py) +
			tex2D<float4>(mipInput, (x + 1.0f) * px, (y + 1.0f) * py) +
			tex2D<float4>(mipInput, (x + 0.0f) * px, (y + 1.0f) * py);

		// Compute the average color.
		color /= 4.0f;
		color *= 255.0f; // Convert to [0,255] range.

		// Clamp values to ensure they do not exceed 255.
		color = fminf(color, make_float4(255.0f));

		// Convert to uchar4 format and write the final pixel to the output surface.
		surf2Dwrite(to_uchar4(color), mipOutput, x * sizeof(uchar4), y);
	}
}

/**
 * @brief Generates a mipmap chain for a given CUDA mipmapped array.
 *
 * This function generates all levels of a mipmapped array by downsampling the higher-level images.
 * Each mipmap level is created by halving the dimensions of the previous level until the dimensions are reduced to 1x1.
 *
 * @param mipmapArray Reference to the CUDA mipmapped array to process.
 * @param size Initial size (extent) of the highest resolution mipmap level.
 */
static void gen_mipmap(hipMipmappedArray_t& mipmapArray, hipExtent size) {
	// Initialize the width and height from the size extent.
	size_t width = size.width;
	size_t height = size.height;

	uint level = 0; // Mipmap level counter.

	// Iterate until the dimensions are reduced to 1x1.
	while (width != 1 || height != 1) {
		// Compute the dimensions of the next mipmap level.
		width = MAX((size_t)1, width / 2);
		height = MAX((size_t)1, height / 2);

		// Retrieve the current and next mipmap levels.
		hipArray_t levelFrom;
		checkCudaErrors(hipGetMipmappedArrayLevel(&levelFrom, mipmapArray, level));
		hipArray_t levelTo;
		checkCudaErrors(hipGetMipmappedArrayLevel(&levelTo, mipmapArray, level + 1));

		// Verify the dimensions of the next level.
		hipExtent levelToSize;
		checkCudaErrors(hipArrayGetInfo(NULL, &levelToSize, NULL, levelTo));
		assert(levelToSize.width == width);
		assert(levelToSize.height == height);
		assert(levelToSize.depth == 0);

		// Create a texture object for reading from the current level.
		hipTextureObject_t texInput;
		hipResourceDesc texResrc = {};
		texResrc.resType = hipResourceTypeArray;
		texResrc.res.array.array = levelFrom;

		hipTextureDesc texDescr = {};
		texDescr.normalizedCoords = 1;
		texDescr.filterMode = hipFilterModeLinear;
		texDescr.addressMode[0] = hipAddressModeClamp;
		texDescr.addressMode[1] = hipAddressModeClamp;
		texDescr.addressMode[2] = hipAddressModeClamp;
		texDescr.readMode = hipReadModeNormalizedFloat;

		checkCudaErrors(hipCreateTextureObject(&texInput, &texResrc, &texDescr, NULL));

		// Create a surface object for writing to the next level.
		hipSurfaceObject_t surfOutput;
		hipResourceDesc surfRes = {};
		surfRes.resType = hipResourceTypeArray;
		surfRes.res.array.array = levelTo;

		checkCudaErrors(hipCreateSurfaceObject(&surfOutput, &surfRes));

		// Configure kernel launch parameters.
		dim3 blockSize(16, 16, 1);
		dim3 gridSize((uint(width) + blockSize.x - 1) / blockSize.x, (uint(height) + blockSize.y - 1) / blockSize.y, 1);

		// Launch the mipmap generation kernel.
		d_gen_mipmap << <gridSize, blockSize >> > (surfOutput, texInput, (uint)width, (uint)height);

		// Synchronize and check for errors.
		checkCudaErrors(hipDeviceSynchronize());
		checkCudaErrors(hipGetLastError());

		// Destroy the surface and texture objects.
		checkCudaErrors(hipDestroySurfaceObject(surfOutput));
		checkCudaErrors(hipDestroyTextureObject(texInput));

		// Increment the mipmap level.
		level++;
	}
}


/**
 * @brief Kernel to sample a mipmapped texture with varying LOD per pixel.
 *
 * This kernel samples a texture using a specified LOD for each pixel, calculates the color data,
 * and writes it to the output buffer.
 *
 * @param texEngine CUDA texture object for the mipmapped texture.
 * @param width Width of the image.
 * @param height Height of the image.
 * @param lod Pointer to an array of LOD values for each pixel.
 * @param dout Output buffer to store the resulting uchar4 color values.
 */
__global__ void d_get_mipmap(
	hipTextureObject_t texEngine,
	const int width,
	const int height,
	const float* lod,
	uchar4* dout
) {
	int xi = blockIdx.x * blockDim.x + threadIdx.x;
	int yi = blockIdx.y * blockDim.y + threadIdx.y;
	int idx = yi * width + xi;

	float u = static_cast<float>(xi) / static_cast<float>(width);
	float v = static_cast<float>(yi) / static_cast<float>(height);
	bool state;

	if (xi < width && yi < height) {
		// Sample the texture with per-pixel LOD.
		float4 data = tex2DLod<float4>(texEngine, u, v, lod[idx], &state);

		// Convert the sampled color to uchar4 and write to the output buffer.
		dout[idx] = to_uchar4(255.0f * data);
	}
}

/**
 * @brief Kernel to sample a mipmapped texture with a uniform LOD.
 *
 * This kernel samples a texture using a single LOD for all pixels, calculates the color data,
 * and writes it to the output buffer.
 *
 * @param texEngine CUDA texture object for the mipmapped texture.
 * @param width Width of the image.
 * @param height Height of the image.
 * @param scale Scale factor used to compute the LOD.
 * @param dout Output buffer to store the resulting uchar4 color values.
 */
__global__ void d_get_mipmap(
	hipTextureObject_t texEngine,
	const int width,
	const int height,
	const float scale,
	uchar4* dout
) {
	int xi = blockIdx.x * blockDim.x + threadIdx.x;
	int yi = blockIdx.y * blockDim.y + threadIdx.y;
	int idx = yi * width + xi;

	float u = (xi + 0.5f) / static_cast<float>(width);
	float v = (yi + 0.5f) / static_cast<float>(height);

	// Calculate the uniform LOD based on the scale.
	float lod = log2(scale);

	if (xi < width && yi < height) {
		// Sample the texture with a uniform LOD.
		float4 data = tex2DLod<float4>(texEngine, u, v, lod);

		// Convert the sampled color to uchar4 and write to the output buffer.
		dout[idx] = to_uchar4(255.0f * data);
	}
}

/**
 * @brief Retrieves a mipmap image with a uniform blur applied using CUDA.
 *
 * This function retrieves the mipmapped image from a CUDA mipmapped array with uniform scaling.
 * It uses texture sampling and stores the result in a device buffer, which is later copied to the host.
 *
 * @param mm_array The CUDA mipmapped array containing the mipmap levels.
 * @param img_size Dimensions of the image and number of mipmap levels (int3: {width, height, n_level}).
 * @param scale Scale factor used to compute the LOD for mipmap sampling.
 * @param dout Host output buffer for storing the resulting uchar4 image.
 */
static void get_mipmap(hipMipmappedArray_t mm_array, const int3 img_size, const float scale, uchar4* dout) {
	const int width = img_size.x;
	const int height = img_size.y;
	const int n_level = img_size.z;
	const int asize = width * height;

	// Initialize texture resource description for mipmapped array.
	hipResourceDesc texResrc = {};
	texResrc.resType = hipResourceTypeMipmappedArray;
	texResrc.res.mipmap.mipmap = mm_array;

	// Initialize texture description.
	hipTextureDesc texDescr = {};
	texDescr.normalizedCoords = 1;
	texDescr.filterMode = button_State[0] ? hipFilterModeLinear : hipFilterModePoint;
	texDescr.mipmapFilterMode = button_State[1] ? hipFilterModeLinear : hipFilterModePoint;
	texDescr.addressMode[0] = hipAddressModeClamp;
	texDescr.addressMode[1] = hipAddressModeClamp;
	texDescr.addressMode[2] = hipAddressModeClamp;
	texDescr.maxMipmapLevelClamp = float(n_level - 1);
	texDescr.readMode = button_State[2] ? hipReadModeNormalizedFloat : hipReadModeElementType;
	texDescr.disableTrilinearOptimization = button_State[3];

	// Create texture object for sampling mipmap levels.
	hipTextureObject_t texEngine;
	checkCudaErrors(hipCreateTextureObject(&texEngine, &texResrc, &texDescr, NULL));

	// Allocate memory for the device output buffer.
	uchar4* d_out;
	checkCudaErrors(hipMalloc(&d_out, asize * sizeof(uchar4)));

	// Define kernel execution parameters.
	dim3 blocksize(16, 16, 1);
	dim3 gridsize((width + blocksize.x - 1) / blocksize.x, (height + blocksize.y - 1) / blocksize.y);

	// Launch the mipmap retrieval kernel.
	d_get_mipmap << <gridsize, blocksize >> > (texEngine, width, height, scale, d_out);

	// Copy the result from the device to the host.
	checkCudaErrors(hipMemcpy(dout, d_out, asize * sizeof(uchar4), hipMemcpyDeviceToHost));
	hipDeviceSynchronize();

	// Cleanup resources.
	checkCudaErrors(hipDestroyTextureObject(texEngine));
	checkCudaErrors(hipFree(d_out));
}

/**
 * @brief Filters an image by generating mipmap levels and retrieving a blurred version.
 *
 * This function generates mipmap levels for a given input image, applies the filtering process,
 * and retrieves the result as a blurred image at a specific scale.
 *
 * @param width Width of the input image.
 * @param height Height of the input image.
 * @param scale Scale factor used for the blur effect.
 * @param src_img Pointer to the input image on the host.
 * @param dst_img Pointer to the output image on the host.
 */
void filter_mipmap(const int width, const int height, const float scale, const uchar4* src_img, uchar4* dst_img) {
	// Calculate the number of mipmap levels based on the largest dimension.
	int n_level = 0;
	int level = max(height, width);
	while (level) {
		level >>= 1;
		n_level++;
	}

	// Define the image extent and channel format.
	hipExtent img_size = { static_cast<size_t>(width), static_cast<size_t>(height), 0 };
	hipChannelFormatDesc ch_desc = hipCreateChannelDesc<uchar4>();

	// Allocate a CUDA mipmapped array.
	hipMipmappedArray_t mm_array;
	checkCudaErrors(hipMallocMipmappedArray(&mm_array, &ch_desc, img_size, n_level));

	// Get the first mipmap level (level 0).
	hipArray_t level0;
	checkCudaErrors(hipGetMipmappedArrayLevel(&level0, mm_array, 0));

	// Copy the input image data to the first mipmap level.
	hipMemcpy3DParms cpy_param = {};
	cpy_param.srcPtr = make_hipPitchedPtr((void*)src_img, width * sizeof(uchar4), width, height);
	cpy_param.dstArray = level0;
	cpy_param.extent = img_size;
	cpy_param.extent.depth = 1;
	cpy_param.kind = hipMemcpyHostToDevice;
	checkCudaErrors(hipMemcpy3D(&cpy_param));

	// Generate mipmap levels from the input image.
	gen_mipmap(mm_array, img_size);

	// Retrieve the filtered mipmap image.
	get_mipmap(mm_array, make_int3(width, height, n_level), scale, dst_img);

	// Free the CUDA mipmapped array.
	checkCudaErrors(hipFreeMipmappedArray(mm_array));
}
